/* Matrix multiplication: C = A * B.
 * Host code.
 */

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <chrono>


#include <hip/hip_runtime.h>

// includes, kernels


#include "assist.h"

#include "matrixmul.h"

int main(int argc, char** argv) {
	bool if_quiet = false;
	unsigned int timer_compute = 0;
	char* matrix_id = NULL;
	char* input_fn = NULL;
	char* gold_fn = NULL;
	int Mw = 0;
	int Mh = 0;
	int Nw = 0;
	int Nh = 0;
	int Pw = 0;
	int Ph = 0;

	if (argc == 2) {
		matrix_id = strdup(argv[1]);
	}
	else {
		fprintf(stderr, "Error: Wrong input parameter numers.\n");
		fprintf(stderr, "Usage:\n"
			"$> ./lab1.1-matrixmul <8, 128, 512, 3072, 4092>\n"
			"Examples:\n "
			" $> ./bab1.1-matrixmul 128\n"
		);
		exit(1);
	}

	Mw = Mh = Mw = Nh = Pw = Ph = atoi(matrix_id);
	input_fn = (char*)malloc(30 * sizeof(char));
	gold_fn = (char*)malloc(30 * sizeof(char));
	sprintf(input_fn, "matrix_%s.bin", matrix_id);
	sprintf(gold_fn, "matrix_%s.gold", matrix_id);
	if (Pw * Ph > 15 * 15) {
		if_quiet = true; // dont show content
	}
	printf("Input matrix size: %d by %d\n", Mw, Mh);

	printf("Setup host side environment:\n");

	printf(" Allocate host memory for matrices M and N.\n");
	printf(" M: %d x %d\n", Mw, Mh);
	printf(" N: %d x %d\n", Nw, Nh);

	unsigned int size_M = Mw * Mh;
	unsigned int mem_size_M = sizeof(float) * size_M;
	float* hostM = (float*)malloc(mem_size_M);
	unsigned int size_N = Nw * Nh;
	unsigned int mem_size_N = sizeof(float) * size_N;
	float* hostN = (float*)malloc(mem_size_N);

	printf(" Allocate memory for the result on the host side.\n");
	unsigned int size_P = Pw * Ph;
	unsigned int mem_size_P = sizeof(float) * size_P;
	float* hostP = (float*)malloc(mem_size_P);

	printf(" Generate input matrix data for matrix M and N.\n");
	GenMatrixFile(input_fn, Pw, Ph, if_quiet);
	unsigned int* matrix = ReadMatrixFile(input_fn, Pw, Ph, true);

	for (int i = 0; i < Mw; i++) {
		for (int j = 0; j < Nw; j++) {
			hostM[i * Mw + j] = hostN[i * Mw + j] = (float)matrix[i * Mw + j];
		}
	}
	free(matrix);
	matrix = NULL;

	printf(" Copmuting matrix multiplication M x N:\n");
	if (Pw*Ph > 512*512) {
		printf(" (It takes time since matrix is larger than 512 by 512\n");
	}

	float* reference = (float*)malloc(mem_size_P);
	computeGold(reference, hostM, hostN, Mh, Mw, Nw);

	printf(" CPU Processing time : %f (ms)\n", 0.0);
	printf(" Matrix data checksum : %g\n", CheckSum(reference, Mw, Nw));

	if (!if_quiet) {
		printf(" matrix data contens:\n");
		printf(" ");
	}

	matrix = (unsigned int*)malloc(Pw * Ph * sizeof(unsigned int));
	for (int i = 0; i < Ph; i++) {
		for (int j = 0; j < Pw; j++) {
			matrix[i * Pw + j] = (unsigned int)reference[i * Pw + j];
			if (!if_quiet) printf("%u ", matrix[i * Pw + j]);
		}
		if (!if_quiet) printf("\n   ");
	}

	if (!if_quiet) printf("\n");

	WriteMatrixFile(gold_fn, matrix, Pw, Ph, 1);
	free(matrix);
	matrix = NULL;

	free(hostM);
	free(hostN);
	free(hostP);

	return 0;
}